#include "../../common/common.h"
#include <hip/hip_runtime.h>
#include <stdio.h>

/*
 * Demonstrate defining the dimensions of a block of threads and a grid of blocks from the host.
 */

int main(int argc, char **argv)
{
    // define total data element
    int nElem = 1024;

		printf("grid\tblock\n");

    // define grid and block structure
    dim3 block (1024);
    dim3 grid  ((nElem + block.x - 1) / block.x);
    printf("%4d\t %4d\n", grid.x, block.x);

    // reset block
    block.x = 512;
    grid.x  = (nElem + block.x - 1) / block.x;
		printf("%4d\t %4d\n", grid.x, block.x);

    // reset block
    block.x = 256;
    grid.x  = (nElem + block.x - 1) / block.x;
		printf("%4d\t %4d\n", grid.x, block.x);

    // reset block
    block.x = 128;
    grid.x  = (nElem + block.x - 1) / block.x;
		printf("%4d\t %4d\n", grid.x, block.x);

    // reset device before you leave
    CHECK(hipDeviceReset());

		getchar();
    return(0);
}

